#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

/// copy from
/// https://github.com/xlite-dev/LeetCUDA/blob/main/kernels/sgemm/sgemm_cublas.cu
/// single precision(float32) GEMM
void cublas_sgemm(float *A, float *B, float *C, int M, int N, int K) {
  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);

  static float alpha = 1.0;
  static float beta = 0.0;

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               HIPBLAS_GEMM_DEFAULT);
}

int main(int argc, char **argv) {

  int M, K, N;
  M = K = N = 4096;
  if (argc == 4) {
    M = atoi(argv[1]);
    K = atoi(argv[2]);
    N = atoi(argv[3]);
  }
  else{
    std::cerr << "Usage: " << argv[0] << " <M> <K> <N>" << std::endl;
    std::cerr << "Default GEMM size: 4096x4096x4096 with single precision(float32)" << std::endl;
  }

  float *h_A = (float *)malloc(M * K * sizeof(float));
  float *h_B = (float *)malloc(K * N * sizeof(float));
  float *h_C = (float *)malloc(M * N * sizeof(float));

  for (int i = 0; i < M * K; i++)
    h_A[i] = 1.0;
  for (int i = 0; i < K * N; i++)
    h_B[i] = 1.0;

  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, M * K * sizeof(float));
  hipMalloc((void **)&d_B, K * N * sizeof(float));
  hipMalloc((void **)&d_C, M * N * sizeof(float));

  // get cublas handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSetMatrix(M, K, sizeof(float), h_A, M, d_A, M);
  hipblasSetMatrix(K, N, sizeof(float), h_B, K, d_B, K);

  // warmup
  std::cout << "Execute single precision(float32) GEMM with " << M << "x" << K << "x" << N << std::endl;
  cublas_sgemm(d_A, d_B, d_C, M, N, K);

  auto start = std::chrono::high_resolution_clock::now();
  const int run_times = 10;
  for (int i = 0; i < run_times; ++i) {
    cublas_sgemm(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();
  }
  auto end = std::chrono::high_resolution_clock::now();

  hipblasGetMatrix(M, N, sizeof(float), d_C, M, h_C, M);

  auto duration =
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  std::cout << "Average time cost: "
            << duration.count() / static_cast<double>(run_times) << " ms"
            << std::endl;
  std::cout << static_cast<double>(2) * M * K * N * run_times /
                   (duration.count() * 1e9)
            << " TFLOPS" << std::endl;

  hipblasDestroy(handle);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
